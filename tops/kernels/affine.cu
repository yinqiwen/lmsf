#include "hip/hip_runtime.h"
#include "cuda_utils.cuh"
#include<stdint.h>

#define AFFINE_OP(TYPENAME, FN_NAME) \
extern "C" __global__ void FN_NAME(  \
    const size_t numel,  \
    const size_t num_dims, \
    const size_t *info, \
    const TYPENAME *inp, \
    TYPENAME *out, \
    const TYPENAME mul, \
    const TYPENAME add \
) {  \
    const size_t *dims = info; \
    const size_t *strides = info + num_dims; \
    if (is_contiguous(num_dims, dims, strides)) { \
        for (unsigned int i = blockIdx.x * blockDim.x + threadIdx.x; i < numel; i += blockDim.x * gridDim.x) { \
            TYPENAME x = inp ? inp[i] : out[i]; \
            out[i] = x * mul + add; \
        } \
    } \
    else { \
        for (unsigned int i = blockIdx.x * blockDim.x + threadIdx.x; i < numel; i += blockDim.x * gridDim.x) { \
            unsigned strided_i = get_strided_index(i, num_dims, dims, strides); \
            TYPENAME x = inp ? inp[strided_i] : out[i]; \
            out[i] = x * mul + add; \
        } \
    } \
} \

#if __CUDA_ARCH__ >= 800
AFFINE_OP(__hip_bfloat16, affine_bf16)
#endif

#if __CUDA_ARCH__ >= 530
AFFINE_OP(__half, affine_f16)
#endif

AFFINE_OP(float, affine_f32)
AFFINE_OP(double, affine_f64)
AFFINE_OP(uint8_t, affine_u8)
AFFINE_OP(uint32_t, affine_u32)
AFFINE_OP(int64_t, affine_i64)
