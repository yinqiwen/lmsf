
#include <hip/hip_runtime.h>
#include<stdint.h>
#include "hip/hip_fp16.h"

template<typename T>
__device__ void fill_with(T *buf, T value, const size_t numel) {
    for (unsigned int i = blockIdx.x * blockDim.x + threadIdx.x; i < numel; i += blockDim.x * gridDim.x) {
        buf[i] = value;
    }
}
extern "C" __global__ void fill_u8(uint8_t *buf, uint8_t value, const size_t numel) { fill_with(buf, value, numel); }
extern "C" __global__ void fill_u32(uint32_t *buf, uint32_t value, const size_t numel) { fill_with(buf, value, numel); }
extern "C" __global__ void fill_i64(int64_t *buf, int64_t value, const size_t numel) { fill_with(buf, value, numel); }
extern "C" __global__ void fill_f16(__half *buf, __half value, const size_t numel) { fill_with(buf, value, numel); }
extern "C" __global__ void fill_f32(float *buf, float value, const size_t numel) { fill_with(buf, value, numel); }
extern "C" __global__ void fill_f64(double *buf, double value, const size_t numel) { fill_with(buf, value, numel); }

#if __CUDA_ARCH__ >= 800
#include <cuda_bf16.h>
extern "C" __global__ void fill_bf16(__nv_bfloat16 *buf, __nv_bfloat16 value, const size_t numel) { fill_with(buf, value, numel); }
#endif
