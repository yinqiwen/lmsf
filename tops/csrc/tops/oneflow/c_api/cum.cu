/*
** BSD 3-Clause License
**
** Copyright (c) 2023, qiyingwang <qiyingwang@tencent.com>, the respective
*contributors, as shown by the AUTHORS file.
** All rights reserved.
**
** Redistribution and use in source and binary forms, with or without
** modification, are permitted provided that the following conditions are met:
** * Redistributions of source code must retain the above copyright notice, this
** list of conditions and the following disclaimer.
**
** * Redistributions in binary form must reproduce the above copyright notice,
** this list of conditions and the following disclaimer in the documentation
** and/or other materials provided with the distribution.
**
** * Neither the name of the copyright holder nor the names of its
** contributors may be used to endorse or promote products derived from
** this software without specific prior written permission.
**
** THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS "AS IS"
** AND ANY EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
** IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR PURPOSE
*ARE
** DISCLAIMED. IN NO EVENT SHALL THE COPYRIGHT HOLDER OR CONTRIBUTORS BE LIABLE
** FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL, EXEMPLARY, OR CONSEQUENTIAL
** DAMAGES (INCLUDING, BUT NOT LIMITED TO, PROCUREMENT OF SUBSTITUTE GOODS OR
** SERVICES; LOSS OF USE, DATA, OR PROFITS; OR BUSINESS INTERRUPTION) HOWEVER
** CAUSED AND ON ANY THEORY OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY,
** OR TORT (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE
** OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
*/
#include <hip/hip_bf16.h>
#include <hip/hip_fp16.h>
#include <hip/hip_runtime_api.h>
#include <memory>
#include <stdexcept>

#include "tops/c_api/c_api.h"
#include "tops/common/err_cuda.h"
#include "tops/oneflow/kernel/cum_forward_kernel.cuh"

extern "C" {
void cuda_cumsum_tensor(CTensorView input, uint32_t dim, hipStream_t stream,
                        CTensorView output) {
  hipDeviceProp_t *prop = getCudaDeviceProp();
  tops::ShapeView in_shape(input);

  switch (input.dtype) {
  case ScalarType::DATA_F16: {
    oneflow::cum_op<half, oneflow::SumFunctor>(
        reinterpret_cast<const half *>(input.ptr), in_shape, nullptr, 0,
        reinterpret_cast<half *>(output.ptr), dim, prop, stream);
    break;
  }
  case ScalarType::DATA_F32: {
    oneflow::cum_op<float, oneflow::SumFunctor>(
        reinterpret_cast<const float *>(input.ptr), in_shape, nullptr, 0,
        reinterpret_cast<float *>(output.ptr), dim, prop, stream);
    break;
  }

  default: {
    throw new std::runtime_error("not supported dtype for cuda_cumsum_tensor");
  }
  }
}
}